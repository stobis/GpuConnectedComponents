#include "hip/hip_runtime.h"
/****************************************************************************************
 *       CONNECTED COMPONENTS ON THE GPU                                                        
 *       ==============================
 *
 *
 *
 *       Copyright (c) 2010 International Institute of Information Technology,
 *       Hyderabad.
 *       All rights reserved.
 *
 *       Permission to use, copy, modify and distribute this software and its
 *       documentation for research purpose is hereby granted without fee,
 *       provided that the above copyright notice and this permission notice appear
 *       in all copies of this software and that you do not sell the software.
 *
 *       THE SOFTWARE IS PROVIDED "AS IS" AND WITHOUT WARRANTY OF ANY KIND,
 *       EXPRESS, IMPLIED OR OTHERWISE.
 *
 *       Please report any issues to Jyothish Soman (jyothish@students.iiit.ac.in)
 *
 *       Please cite following paper, if you use this software for research purpose
 *
 *       "Fast GPU Algorithms for Graph Connectivity, Jyothish Soman, K. Kothapalli, 
 *       and P. J. Narayanan, in Proc. of Large Scale Parallel Processing, 
 *       IPDPS Workshops, 2010.
 *
 *
 *
 *
 *       Created by Jyothish Soman
 *											
 ****************************************************************************************/

#include<hip/hip_runtime_api.h>
#include<helper_functions.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>

#include "conn.cuh"

namespace cc {

/*
 *
 *    Function to speedup the selection process in the first iteration
 *    The ancestor tree is initialized to the add the edge from larger edge to its smaller neighbour in this method.
 *    The process is random and each edge performs this task independently.
 *    select_winner_init
 *
 */

__global__ 
void select_winner_init(int* an,edge *ed_list,int num_e,int num_n,int*flag,char*mark,int*win_edge){
    int a,b,x,y,mn,mx;
    long long int t;
    a=blockIdx.y*gridDim.x+blockIdx.x;
    b=threadIdx.x;
    a=a*512+b;
    if(a<num_e){
        t=ed_list[a].x;
        x=(int)t & 0xFFFFFFFF;
        y=(int)(t>>32);

        mx=x>y?x:y;
        mn=x+y-mx;
        // an[mx]=mn;
        win_edge[mx]=a;
    }
    return;
}

__global__ void select_tree_edges_and_merge_init(int *an, edge *ed_list,
                                            int num_e, int num_n, int *flag,
                                            char *mark, int *win_edge, int *is_tree) {
    int a, b, x, y, a_x, a_y, mn, mx;
    long long int t;
    a = blockIdx.y * gridDim.x + blockIdx.x;
    b = threadIdx.x;
    a = a * 512 + b;

    if (a < num_n) {
        if (win_edge[a] != -1) {
            is_tree[win_edge[a]] = 1;

            t=ed_list[win_edge[a]].x;
            x=(int)t & 0xFFFFFFFF;
            y=(int)(t>>32);

            mx=x>y?x:y;
            mn=x+y-mx;
            an[mx]=mn;
        }
    }
}

/*
   Function to hook from higher valued tree to lower valued tree. For details, read the PPL Paper or LSPP paper or my master's thesis.
   Following greener's algorithm, there are two iterations, one from lower valued edges to higher values edges
   and the second iteration goes vice versa. The performance of this is largely related to the input.

 */   



__global__ 
void select_winner2(int* an,edge *ed_list,int num_e,int num_n,int*flag,char*mark,int*win_edge){
    int a,b,x,y,a_x,a_y,mn,mx;
    long long int t;
    a=blockIdx.y*gridDim.x+blockIdx.x;
    b=threadIdx.x;
    __shared__ int s_flag;
    a=a*512+b;
    if(b==1)
        s_flag=0;
    __syncthreads();
    if(a<num_e){
        if(mark[a]==0){
            t=ed_list[a].x;
            x=(int)t & 0xFFFFFFFF;
            y=(int)(t>>32);

            a_x=an[x];
            a_y=an[y];
            mx=a_x>a_y?a_x:a_y;
            mn=a_x+a_y-mx;
            if(mn==mx){
                mark[a]=-1;
            }
            else{
                // an[mn]=mx;
                win_edge[mn]=a;
                s_flag=1;
            }
        }
    }
    __syncthreads();
    if(b==1){
        if(s_flag==1){
            *flag=1;
        }
    }
    return;
}

__global__ void select_tree_edges_and_merge2(int *an, edge *ed_list,
                                            int num_e, int num_n, int *flag,
                                            char *mark, int *win_edge, int *is_tree) {
    int a, b, x, y, a_x, a_y, mn, mx;
    long long int t;
    a = blockIdx.y * gridDim.x + blockIdx.x;
    b = threadIdx.x;
    a = a * 512 + b;

    if (a < num_n) {
        if (win_edge[a] != -1) {
            is_tree[win_edge[a]] = 1;

            t = ed_list[win_edge[a]].x;
            x = (int)t & 0xFFFFFFFF;
            y = (int)(t >> 32);

            a_x = an[x];
            a_y = an[y];
            mx = a_x > a_y ? a_x : a_y;
            mn = a_x + a_y - mx;

            an[mn] = mx;
        }
    }
}

/*
   Function to hook from lower valued to higher valued trees. 



 */   
__global__ 
void select_winner(int* an,edge *ed_list,int num_e,int num_n,int*flag,char*mark,int*win_edge){
    int a,b,x,y,a_x,a_y,mn,mx;
    long long int t;
    a=blockIdx.y*gridDim.x+blockIdx.x;
    b=threadIdx.x;
    __shared__ int s_flag;
    a=a*512+b;
    if(b==1)
        s_flag=0;
    __syncthreads();
    if(a<num_e){
        if(mark[a]==0){
            t=ed_list[a].x;
            x=(int)t & 0xFFFFFFFF;
            y=(int)(t>>32);

            a_x=an[x];
            a_y=an[y];
            mx=a_x>a_y?a_x:a_y;
            mn=a_x+a_y-mx;
            if(mn==mx){
                mark[a]=-1;
            }
            else{
                // an[mx]=mn;
                win_edge[mx]=a;
                s_flag=1;
            }
        }
    }
    __syncthreads();
    if(b==1){
        if(s_flag==1){
            *flag=1;
        }
    }
    return;
}

__global__ void select_tree_edges_and_merge(int *an, edge *ed_list,
                                            int num_e, int num_n, int *flag,
                                            char *mark, int *win_edge, int *is_tree) {
    int a, b, x, y, a_x, a_y, mn, mx;
    long long int t;
    a = blockIdx.y * gridDim.x + blockIdx.x;
    b = threadIdx.x;
    a = a * 512 + b;

    if (a < num_n) {
        if (win_edge[a] != -1) {
            is_tree[win_edge[a]] = 1;

            t = ed_list[win_edge[a]].x;
            x = (int)t & 0xFFFFFFFF;
            y = (int)(t >> 32);

            a_x = an[x];
            a_y = an[y];
            mx = a_x > a_y ? a_x : a_y;
            mn = a_x + a_y - mx;

            an[mx] = mn;
        }
    }
}

__global__ 
void p_jump(int num_n,int* an,int *flag){
    int a,b,x,y;
    a=blockIdx.y*gridDim.x+blockIdx.x;		
    b=threadIdx.x;
    a=a*512+b;
    __shared__ int s_f;
    if(a>=num_n)
        return;
    if(b==1){
        s_f=0;
    }
    __syncthreads();
    if(a<num_n){
        y=an[a];
        x=an[y];
        if(x!=y){
            s_f=1;
            an[a]=x;
        }
    }
    if(b==1){
        if(s_f==1){
            *flag=1;
        }
    }
}


/*
   Function to do a masked jump
   Nodes are either root nodes or leaf nodes. Leaf nodes are directly connected to the root nodes, hence do not
   need to jump itertively. Once root nodes have reascertained the new root nodes, the leaf nodes can just jump once


 */
__global__ 
void p_jump_masked(int num_n,int* an,int *flag,char*mask){
    int a,b,x,y;
    a=blockIdx.y*gridDim.x+blockIdx.x;
    b=threadIdx.x;
    a=a*512+b;
    __shared__ int s_f;
    if(a>=num_n)
        return;
    if(b==1){
        s_f=0;
    }

    __syncthreads();
    if(mask[a]==0){
        y=an[a];
        x=an[y];
        if(x!=y){
            s_f=1;
            an[a]=x;
        }
        else{
            mask[a]=-1;
        }
    }
    if(b==1){
        if(s_f==1){
            *flag=1;
        }
    }
}

/*

   Function for pointer jumping in the tree, the tree height is shortened by this method.
   Here the assumption is that all the nodes are root nodes, or not known whether they are leaf nodes.
   Works well in the early iterations

 */

__global__ 
void p_jump_unmasked(int num_n,int* an,char *mask){
    int a,b,x,y;
    a=blockIdx.y*gridDim.x+blockIdx.x;
    b=threadIdx.x;
    a=a*512+b;
    if(a>=num_n)
        return;
    __syncthreads();
    if(mask[a]==1){
        y=an[a];
        x=an[y];
        an[a]=x;
    }
}

/*
   Function to create self pointing tree.
 */
__global__ 
void update_an(int*an,int num_n){
    int a,b;
    a=blockIdx.y*gridDim.x+blockIdx.x;		
    b=threadIdx.x;
    a=a*512+b;
    if(a>=num_n)
        return;
    an[a]=a;

    return;
}

/*
   Function to initialize each edge as a clean copy. 
 */
__global__ 
void	update_mark(char *mark,int num_e){
    int j;
    j=blockIdx.y*gridDim.x+blockIdx.x;
    j=j*512+threadIdx.x;
    if(j>=num_e)
        return;
    mark[j]=0;
}

/*
   Function to check if each node is the parent of itself or not and to update it as a leaf or root node

 */

__global__ 
void update_mask(char *mask,int n,int *an){
    int j;
    j=blockIdx.y*gridDim.x+blockIdx.x;
    j=j*512+threadIdx.x;
    if(j>=n)
        return;
    mask[j]=an[j]==j?0:1;
    return;
}

void compute(int const num_n, int const num_e, edge *d_ed_list, int * d_is_tree) 
{
    // findCudaDevice(argc,(const char**) argv);

    // edge* ed_list;
    int nnx,nny,nex,ney;	
//    unsigned int timer1 = 0;
//    checkCudaErrors( cutCreateTimer( &timer1));
//    checkCudaErrors( cutStartTimer( timer1));


    // load_graph(&ed_list,&num_n,&num_e);

    int flag,*d_winner,*d_an;
    int *d_flag,*an;
    char*d_mark,*mark;
    char*mask;


    int num_threads,num_blocks_n,num_blocks_e;
    num_threads=512;
    num_blocks_n=(num_n/512)+1;
    num_blocks_e=(num_e/512)+1;
    nny=(num_blocks_n/1000)+1;
    nnx=1000;
    nex=(num_blocks_e/1000)+1;
    ney=1000;
    dim3  grid_n( nnx, nny);
    dim3  grid_e( nex, ney);
    dim3  threads( num_threads, 1);

    an=(int*)calloc(num_n,sizeof(int));
    checkCudaErrors(hipMalloc((void**)&d_mark,num_e*sizeof(char)));
    checkCudaErrors(hipMalloc((void**)&mask,num_e*sizeof(char)));
    checkCudaErrors(hipMalloc((void**)&d_winner,num_n*sizeof(int)));
    checkCudaErrors(hipMalloc((void**)&d_an,num_n*sizeof(int)));
    // checkCudaErrors(hipMalloc((void**)&d_ed_list,num_e*sizeof(edge)));
    checkCudaErrors(hipMalloc((void**)&d_flag,sizeof(int)));
    // checkCudaErrors(hipMemcpy(d_ed_list,ed_list,num_e*sizeof(edge),hipMemcpyHostToDevice));

    int *d_win_edge;
    checkCudaErrors(hipMalloc((void**)&d_win_edge,num_n*sizeof(int)));
    checkCudaErrors(hipMemset(d_win_edge, -1, num_n * sizeof(int)));
        

    //   Finished intializing space for the program, ideally timing should be from here.

    clock_t t = clock();
    


    update_mark<<< grid_e,threads>>>(d_mark,num_e);
    update_an<<< grid_n,threads>>>(d_an,num_n);
    hipDeviceSynchronize();

    hipDeviceSynchronize();


    //First round of select winner


    select_winner_init<<< grid_e,threads>>>(d_an,d_ed_list,num_e,num_n,d_flag,d_mark,d_win_edge);
    hipDeviceSynchronize();

    select_tree_edges_and_merge_init<<<grid_n, threads>>>(
        d_an, d_ed_list, num_e, num_n, d_flag, d_mark, d_win_edge, d_is_tree);
    
    hipDeviceSynchronize();

//    CUT_CHECK_ERROR("Kernel execution failed");


    do{
        flag=0;
        checkCudaErrors(hipMemcpy(d_flag,&flag,sizeof(int),hipMemcpyHostToDevice));
        p_jump<<< grid_n,threads>>>(num_n,d_an,d_flag);
        hipDeviceSynchronize();

//        CUT_CHECK_ERROR("Kernel execution failed");
        checkCudaErrors(hipMemcpy(&flag,d_flag,sizeof(int),hipMemcpyDeviceToHost));
    }while(flag);

    //main code starts
    //
    update_mask<<< grid_n,threads>>>(mask,num_n,d_an);
    int lpc=1;
    do{
        checkCudaErrors(hipMemset(d_win_edge, -1, num_n * sizeof(int)));
    
        flag=0;				
        checkCudaErrors(hipMemcpy(d_flag,&flag,sizeof(int),hipMemcpyHostToDevice));
        if(lpc!=0){
            select_winner<<< grid_e,threads>>>(d_an,d_ed_list,num_e,num_n,d_flag,d_mark,d_win_edge);
            hipDeviceSynchronize();

            select_tree_edges_and_merge<<<grid_n, threads>>>(
                d_an, d_ed_list, num_e, num_n, d_flag, d_mark, d_win_edge, d_is_tree);

            lpc++;
            lpc=lpc%4;
        }
        else{

            select_winner2<<< grid_e,threads>>>(d_an,d_ed_list,num_e,num_n,d_flag,d_mark,d_win_edge);
            hipDeviceSynchronize();

            select_tree_edges_and_merge2<<<grid_n, threads>>>(
                d_an, d_ed_list, num_e, num_n, d_flag, d_mark, d_win_edge, d_is_tree);

            lpc=0;
        }
        hipDeviceSynchronize();

 //       CUT_CHECK_ERROR("Kernel execution failed");
        checkCudaErrors(hipMemcpy(&flag,d_flag,sizeof(int),hipMemcpyDeviceToHost));
        if(flag==0){
            break;
        }

//        CUT_CHECK_ERROR("Kernel execution failed");

        int flg;
        do{
            flg=0;
            checkCudaErrors(hipMemcpy(d_flag,&flg,sizeof(int),hipMemcpyHostToDevice));
            p_jump_masked<<< grid_n,threads>>>(num_n,d_an,d_flag,mask);
            hipDeviceSynchronize();

//            CUT_CHECK_ERROR("Kernel execution failed");
            checkCudaErrors(hipMemcpy(&flg,d_flag,sizeof(int),hipMemcpyDeviceToHost));
        }while(flg);

        p_jump_unmasked<<< grid_n,threads>>>(num_n,d_an,mask);
        hipDeviceSynchronize();
//        CUT_CHECK_ERROR("Kernel execution failed");

        update_mask<<< grid_n,threads>>>(mask,num_n,d_an);
//        CUT_CHECK_ERROR("Kernel execution failed");
        hipDeviceSynchronize();
    }while(flag);
    t = clock() - t;
    /* checkCudaErrors( cutStopTimer( timer)); */
    /* printf( "%f\n", cutGetTimerValue( timer)); */
    /* checkCudaErrors( cutDeleteTimer( timer)); */
    printf ("Time required for computing connected components on the graph is: %f seconds.\n",((float)t)/CLOCKS_PER_SEC);
    
    
    mark=(char*)calloc(num_e,sizeof(char));
    //end of main loop
    checkCudaErrors(hipMemcpy(an,d_an,num_n*sizeof(int),hipMemcpyDeviceToHost));
    int j,cnt=0;
    for(j=0;j<num_n;j++){
        // printf("anj %d %d\n", j, an[j]);
        if(an[j]==j){
            cnt++;
        }
    }

    printf("The number of components=%d\n",cnt);
    free(an);
    free(mark);
    checkCudaErrors(hipFree(d_an));
    // checkCudaErrors(hipFree(d_ed_list));
    checkCudaErrors(hipFree(d_flag));
    checkCudaErrors(hipFree(d_mark));
}

} // namespace cc
